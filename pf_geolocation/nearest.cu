// brute force nearest neighbor
//original code from: http://nghiaho.com/?p=416

#include <hip/hip_runtime.h>
#include <float.h>
#define CUDA_NN_DIM 2 // data dimension

__global__ void nearest(const float *query_x, const float *query_y, int query_pts, const float *data, int data_pts,
                    int *idxs)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float dist_sq;

    if(idx >= query_pts)
        return;

    int best_idx = -1;
    float best_dist = FLT_MAX;

    for(int i=0; i < data_pts; i++) {
        dist_sq = 0;

        
        float d = query_x[idx] - data[i*CUDA_NN_DIM];
        dist_sq += d*d;
        d = query_y[idx] - data[i*CUDA_NN_DIM+1];
        dist_sq += d*d;
        

        if(dist_sq < best_dist) {
            best_dist = dist_sq;
            best_idx = i;
        }
    }

    idxs[idx] = best_idx;
//    dist_sq[idx] = best_dist;
}
